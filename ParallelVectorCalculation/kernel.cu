﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void SingleBlockAdd(double* _pt1, double* _pt2, double* _r) {
    int tID = threadIdx.x;
    _r[tID] = _pt1[tID] + _pt2[tID];
}

__global__ void MultipleBlockAdd(double* _pt1, double* _pt2, double* _r) {
    int tID = blockIdx.x * blockDim.x + threadIdx.x;
    _r[tID] = _pt1[tID] + _pt2[tID];
}

#define EXPORTED_METHOD extern "C" __declspec(dllexport) 

EXPORTED_METHOD
void VectorAdd(double* point1, double* point2, int len, double* result)
{
    double* dpt1;
    double* dpt2; 
    double* dresult;

    int memSize = sizeof(double) * len * 3;

    hipMalloc(&dpt1, memSize); hipMemset(dpt1, 0, memSize);
    hipMalloc(&dpt2, memSize); hipMemset(dpt2, 0, memSize);
    hipMalloc(&dresult, memSize); hipMemset(dresult, 0, memSize);

    hipMemcpy(dpt1, point1, memSize, hipMemcpyHostToDevice);
    hipMemcpy(dpt2, point2, memSize, hipMemcpyHostToDevice);

    SingleBlockAdd <<<1, len * 3 >>> (dpt1, dpt2, dresult);
    
    hipMemcpy(result, dresult, memSize, hipMemcpyDeviceToHost);

    hipFree(dpt1);
    hipFree(dpt2);
    hipFree(dresult);    
}

EXPORTED_METHOD
void BlockVectorAdd(double* point1, double* point2, int len, double* result)
{
    double* dpt1;
    double* dpt2; 
    double* dresult;

    int memSize = sizeof(double) * len * 3;

    hipMalloc(&dpt1, memSize); hipMemset(dpt1, 0, memSize);
    hipMalloc(&dpt2, memSize); hipMemset(dpt2, 0, memSize);
    hipMalloc(&dresult, memSize); hipMemset(dresult, 0, memSize);

    hipMemcpy(dpt1, point1, memSize, hipMemcpyHostToDevice);
    hipMemcpy(dpt2, point2, memSize, hipMemcpyHostToDevice);

    MultipleBlockAdd <<<ceil((len*3)/1024), 1024>>> (dpt1, dpt2, dresult);
    
    hipMemcpy(result, dresult, memSize, hipMemcpyDeviceToHost);

    hipFree(dpt1);
    hipFree(dpt2);
    hipFree(dresult);  

}

